#include <iostream>
#include <string.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "plane_2d.h"

using namespace std;

namespace popart {

__host__
void* PlaneBase::allocDev2D( size_t& pitch, int w, int h, int elemSize )
{
    void*       ptr;
    hipError_t err;
    err = hipMallocPitch( &ptr, &pitch, w * elemSize, h );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Cannot allocate " << w*h*elemSize << " bytes of CUDA memory" << endl
             << "    Cause: " << hipGetErrorString( err ) << endl;
        exit( -1 );
    }
    return ptr;
}

__host__
void PlaneBase::freeDev2D( void* data )
{
    hipError_t err;
    err = hipFree( data );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to free CUDA memory." << endl
             << "    Cause: " << hipGetErrorString( err ) << endl;
        exit( -1 );
    }
}

__host__
void* PlaneBase::allocHost2D( int w, int h, int elemSize, PlaneMapMode m )
{
    int sz = w * h * elemSize;

    if( m == Unaligned ) {
        void* ptr = malloc( sz );

        if( ptr != 0 ) return ptr;
        
        char buf[100];
        strerror_r( errno, buf, 100 );
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to allocate " << sz << " bytes of unaligned host memory." << endl
             << "    Cause: " << buf << endl;
        exit( -1 );
    } else if( m == PageAligned ) {
        void* ptr;
        long  pagesize = sysconf(_SC_PAGESIZE);
        int   retval = posix_memalign( &ptr, pagesize, sz );

        if( retval == 0 ) return ptr;

        char buf[100];
        strerror_r( errno, buf, 100 );
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to allocate " << sz << " bytes of page-aligned host memory." << endl
             << "    Cause: " << buf << endl
             << "    Trying to allocate unaligned instead." << endl;

        return allocHost2D( w, h, elemSize, Unaligned );
    } else if( m == CudaAllocated ) {
        void* ptr;
        hipError_t err;
        err = hipHostMalloc( &ptr, sz );

        if( err == hipSuccess ) return ptr;

        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to allocate " << sz << " bytes of aligned and pinned host memory." << endl
             << "    Cause: " << hipGetErrorString(err) << endl
             << "    Trying to allocate unaligned instead." << endl;

        return allocHost2D( w, h, elemSize, Unaligned );
    } else {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Alignment not correctly specified in host plane allocation" << endl;
        exit( -1 );
    }
}

__host__
void PlaneBase::freeHost2D( void* data, PlaneMapMode m )
{
    if( data != 0 ) {
        if( m != CudaAllocated ) {
            free( data );
        } else {
            hipHostFree( data );
        }
        data = 0;
    }
}

__host__
void PlaneBase::memcpyToDevice( void* dst, int dst_pitch,
                                void* src, int src_pitch,
                                short cols, short rows,
                                int elemSize )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2D( dst, dst_pitch,
                        src, src_pitch,
                        cols*elemSize, rows,
                        hipMemcpyHostToDevice );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to copy 2D plane host-to-device." << endl
             << "    Cause: " << hipGetErrorString( err ) << endl;
        exit( -1 );
    }
}

__host__
void PlaneBase::memcpyToDevice( void* dst, int dst_pitch,
                                void* src, int src_pitch,
                                short cols, short rows,
                                int elemSize,
                                hipStream_t stream )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2DAsync( dst, dst_pitch,
                             src, src_pitch,
                             cols*elemSize, rows,
                             hipMemcpyHostToDevice,
                             stream );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to copy 2D plane host-to-device." << endl
             << "    Cause: " << hipGetErrorString( err ) << endl;
        exit( -1 );
    }
}

__host__
void PlaneBase::memcpyToHost( void* dst, int dst_pitch,
                              void* src, int src_pitch,
                              short cols, short rows,
                              int elemSize )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2D( dst, dst_pitch,
                        src, src_pitch,
                        cols*elemSize, rows,
                        hipMemcpyDeviceToHost );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to copy 2D plane device-to-host." << endl
             << "    Cause: " << hipGetErrorString( err ) << endl;
        exit( -1 );
    }
}

__host__
void PlaneBase::memcpyToHost( void* dst, int dst_pitch,
                              void* src, int src_pitch,
                              short cols, short rows,
                              int elemSize,
                              hipStream_t stream )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2DAsync( dst, dst_pitch,
                             src, src_pitch,
                             cols*elemSize, rows,
                             hipMemcpyDeviceToHost,
                             stream );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to copy 2D plane device-to-host." << endl
             << "    Cause: " << hipGetErrorString( err ) << endl;
        exit( -1 );
    }
}

#ifdef PLANE2D_CUDA_OP_DEBUG
__host__
void PlaneBase::waitAndCheck( hipStream_t stream ) const
{
    hipStreamSynchronize( stream );
    hipError_t err = hipGetLastError( );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed in error check after async 2D plane operation." << endl
             << "    Cause: " << hipGetErrorString( err ) << endl;
        exit( -1 );
    }
}
#endif // PLANE2D_CUDA_OP_DEBUG

} // namespace popart

